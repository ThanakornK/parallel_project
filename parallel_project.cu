#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <string>
#include <sstream>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define DocNum 10
#define Doc_Size 9
#define Classes 2
#define BLOCK_SIZE 512
#define DocWords 20
#define DocClass_0 6
#define DocClass_1 4

using namespace std;
using namespace thrust;

__host__ int isin(host_vector<string> vocab, string f) // just check that is string f in vector vocab?
{

    // cout << "debug " << f << endl;
    if (!vocab.empty())
    {
        // cout << "vocab not empty" << endl;
        for (int i = 0; i < vocab.size(); i++)
        {

            if (vocab[i].compare(f) == 0)
            {

                return i;
            }
        }
    }

    return -1;
};

__host__ void translateDoc( host_vector<string> vocabList,host_vector<string> docs, int* docWord_arr) {
    
    int index = 0;
    for (int i = 0; i < docs.size(); i++) {
        stringstream ssin(docs[i]);

        string word;
        while (ssin >> word)
        {
            docWord_arr[index] = isin(vocabList, word);
            index++;
        }
    }
    

   
}

__host__ void getVocab(host_vector<string> &docList, host_vector<string> &vocabList) {

    for (int i = 0; i < docList.size(); i++) {
        stringstream ssin(docList[i]);

        string word;
        // printf("%s\n", word);
        while (ssin >> word) {
            if (isin(vocabList, word) == -1){
                vocabList.push_back(word);
            }
        }
    }
    // for (int i = 0; i < DocNum; i++)
    // {

    //     stringstream ssin(docList[i]);

    //     string word;
    //     printf("%s\n", word);
    //     while (ssin >> word)
    //     {

    //         if (isin(vocabList, word) == -1)
    //         {

    //             vocabList.push_back(word);
    //         }
    //     }
    // }
}

__global__ void term_ClassN(int * doc, int * termInClass, int nDoc) {
    int tid = threadIdx.x;

    // printf("this is from term_ClassN thread %d\n", tid);

    for (int j = 0; j < nDoc*DocWords; j++) {
        
        if (tid == doc[j]) {
            // printf("thread id %d and doc word is %d\n",tid, doc[j]);
            termInClass[tid] = termInClass[tid] + 1;
        }
    }

}

__global__ void find_posterior(int * termInClass, int * nDoc_class, double * posteriorProb) {
    int tid = threadIdx.x;

    double pos = ((termInClass[tid] + 1) * 1.0) / ((*nDoc_class + 2) * 1.0);
    // printf("this is thread %d and pos is %lf add arr index %d\n",tid,pos,tid * (*cur_class));
    posteriorProb[tid] = pos;
    
}

int main() {

    // class 0 is ads class 1 is not ads

    host_vector<string> c_0;
    host_vector<string> c_1;

    c_0.push_back("eligator hosting server we have hosting that can serve you Just paid 20 dollars per month for hosting your web");
    c_0.push_back("explore our selection of local favorites with 0 dollars delivery fee for your first month 10 dollars order minimum terms");
    c_0.push_back("need graphic design help in just a few clicks you can scale your creative output by hiring our pro designer");
    c_0.push_back("so your business is up and running now what grow with a marketing crm that gets smarter as you go");
    c_0.push_back("start and grow your business with shopify turn what you love into what you sell try shopify for free today");
    c_0.push_back("looking for new glasses answer a few quick questions and we will suggest some great looking frames for you free");


    c_1.push_back("today I feel like I want to sleep all day I just wanna lay in my bed and go sleep");
    c_1.push_back("this week is rainy everyday I have to take my umbrella everyday it make me annoy sometimes when I walk");
    c_1.push_back("I am so tired I just want to rest in my vacation time go see outside not sit in table");
    c_1.push_back("she go to market to buy some pills but when she went out she forgot her wallet at her home");


    host_vector<string> vocabList;
    double priorProb[Classes];

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    priorProb[0] = ((DocClass_0 + 1) * 1.0) / (((DocClass_0 + DocClass_1) + 2) * 1.0);
    priorProb[1] = ((DocClass_1 + 1) * 1.0) / (((DocClass_0 + DocClass_1) + 2) * 1.0);

    getVocab(c_0, vocabList);
    getVocab(c_1, vocabList);

    int class_0_arr[DocClass_0*DocWords];
    int class_1_arr[DocClass_1*DocWords];

    int termInClass_0[DocNum*DocWords];
    int termInClass_1[DocNum*DocWords];

    for (int t = 0; t < DocNum*DocWords; t++) {    // set value in termInClass to 0 for count in function
        termInClass_0[t] = 0;
        termInClass_1[t] = 0;
    }

    translateDoc(vocabList, c_0, class_0_arr);
    translateDoc(vocabList, c_1, class_1_arr);

    // kernel ---------------------------------------------------
    int * d_doc_array, *d_termInClass_0,*d_termInClass_1 ;


    // class 0

    hipMalloc((void **) &d_doc_array, DocClass_0*DocWords*sizeof(int));
    hipMalloc((void **) &d_termInClass_0, DocNum*DocWords*sizeof(int));
            
    hipMemcpy(d_doc_array, &class_0_arr, DocClass_0*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_termInClass_0, &termInClass_0, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(start);

    term_ClassN<<<1,vocabList.size()>>>(d_doc_array, d_termInClass_0,DocClass_0);

    hipMemcpy(&termInClass_0, d_termInClass_0, DocNum*DocWords*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_doc_array);
    hipFree(d_termInClass_0);

    // ---------------

    // class 1

    hipMalloc((void **) &d_doc_array, DocClass_1*DocWords*sizeof(int));
    hipMalloc((void **) &d_termInClass_1, DocNum*DocWords*sizeof(int));
            
    hipMemcpy(d_doc_array, &class_1_arr, DocClass_1*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_termInClass_1, &termInClass_1, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);

    term_ClassN<<<1,vocabList.size()>>>(d_doc_array, d_termInClass_1,DocClass_1);

    hipMemcpy(&termInClass_1, d_termInClass_1, DocNum*DocWords*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_doc_array);
    hipFree(d_termInClass_1);


    int * d_nDoc_class ;
    
    double * d_posteriorProb_class0, *d_posteriorProb_class1;

    double posteriorProb_class0[DocWords*DocNum];
    double posteriorProb_class1[DocWords*DocNum];

    // posteriorProb class 0 ---------------------

    int size_of_docClass = DocClass_0;

    hipMalloc((void **) &d_termInClass_0, DocNum*DocWords*sizeof(int));
    hipMalloc((void **) &d_nDoc_class, sizeof(int));
    hipMalloc((void **) &d_posteriorProb_class0, (DocWords*DocNum)*sizeof(double));

    hipMemcpy(d_termInClass_0, &termInClass_0, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nDoc_class, &size_of_docClass, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_posteriorProb_class0, &posteriorProb_class0, (Classes*DocWords*DocNum)*sizeof(double), hipMemcpyHostToDevice);

    find_posterior<<<1,vocabList.size()>>>(d_termInClass_0, d_nDoc_class, d_posteriorProb_class0);

    hipMemcpy(&posteriorProb_class0, d_posteriorProb_class0, (DocWords*DocNum)*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_termInClass_0);
    hipFree(d_nDoc_class);
    hipFree(d_posteriorProb_class0);
        
    // -------------------------------------------

    // cout << "----------" << endl;

    // class 1 -----------------------------------

    size_of_docClass = DocClass_1;

    hipMalloc((void **) &d_termInClass_1, DocNum*DocWords*sizeof(int));
    hipMalloc((void **) &d_nDoc_class, sizeof(int));
    hipMalloc((void **) &d_posteriorProb_class1, (DocWords*DocNum)*sizeof(double));

    hipMemcpy(d_termInClass_1, &termInClass_1, DocNum*DocWords*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nDoc_class, &size_of_docClass, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_posteriorProb_class1, &posteriorProb_class1, (DocWords*DocNum)*sizeof(double), hipMemcpyHostToDevice);

    find_posterior<<<1,vocabList.size()>>>(d_termInClass_1, d_nDoc_class, d_posteriorProb_class1);

    hipEventRecord(stop);
	hipEventSynchronize(stop);

    float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(&posteriorProb_class1, d_posteriorProb_class1, (DocWords*DocNum)*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_termInClass_1);
    hipFree(d_nDoc_class);
    hipFree(d_posteriorProb_class1);

    // --------------------------------------------

    // show value of priorProb and posteriorProb

    cout << endl <<"This is priorProb" << endl << endl;

    for (int pp = 0 ; pp < Classes; pp++) {
        cout << priorProb[pp] << endl;
    }

    cout << endl << "this is posteriorProb" << endl << endl;

    cout << "Class 0" << endl << endl;
    
    for (int pp0 = 0; pp0 < vocabList.size(); pp0++) {
        cout << posteriorProb_class0[pp0] << endl;
    }

    cout << endl << "Class 1" << endl << endl;

    for (int pp1 = 0; pp1 < vocabList.size(); pp1++) {
        cout << posteriorProb_class1[pp1] << endl;
    }

    // -----------------------------------------

    cout << endl << "Time used: " << milliseconds << " milliseconds\n" << endl;;

    
}